#include "hip/hip_runtime.h"
/*
Copyright (C) 2017 the team of Jim00000, ActKz and pityYo

Permission is hereby granted, free of charge, to any person obtaining a copy of this software 
and associated documentation files (the "Software"), to deal in the Software without restriction, 
including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or substantial 
portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT 
LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. 
IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE 
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

/**
 * @file        wave_equation_cuda_api.cu
 * @author      Jim00000
 * @date        12.8.2017
 */

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include "wave_equation_cuda_api.h"

__global__ void _cuda_update_kernal_(double* d_data, double* d_olddata, double* d_newdata, int row_size, int col_size, double C, double K, double dt);

__global__ void _cuda_update_kernal_(double* d_data, double* d_olddata, double* d_newdata, int row_size, int col_size, double C, double K, double dt)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    const int i = idy, j = idx;

    double potential = d_data[(i + 1) * col_size + j] + d_data[(i - 1) * col_size + j] + d_data[i * col_size + j + 1] +
    d_data[i * col_size + j - 1] - 4 * d_data[i * col_size + j];
    
    d_newdata[i * col_size + j] = ( pow(C * dt, 2) * potential * 2 + 4 * d_data[i * col_size + j] - d_olddata[i * col_size + j] *
    (2 - K * dt) ) / (2 + K * dt);
}

void c_cuda_update(double* data, double* olddata, double* newdata, int row_size, int col_size, double C, double K, double dt)
{
    const static int ARRAY_SIZE = row_size * col_size;
    const static int ARRAY_BYTES = ARRAY_SIZE * sizeof(double);     
    static bool initialized = false;

    // Declare GPU memory pointers
    static double* d_data;
    static double* d_olddata;
    static double* d_newdata;

    if(initialized == false) {
        initialized = true;
        // Allocate GPU memory
        hipMalloc((void**) &d_data, ARRAY_BYTES);
        hipMalloc((void**) &d_olddata, ARRAY_BYTES);
        hipMalloc((void**) &d_newdata, ARRAY_BYTES);
    }


    // Transfer memory to GPU memory
    hipMemcpy(d_data, data, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_olddata, olddata, ARRAY_BYTES, hipMemcpyHostToDevice);

    const static int THREADS_COUNT = 16;
    dim3 threads(THREADS_COUNT, THREADS_COUNT);
    dim3 blocks(col_size / threads.x + 1, row_size / threads.y + 1);

    // Launch the kernel
    _cuda_update_kernal_<<<blocks, threads>>>(d_data, d_olddata, d_newdata, row_size, col_size, C, K, dt);
    hipDeviceSynchronize();

    // Transfer GPU memory to host memory
    hipMemcpy(data, d_data, ARRAY_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(newdata, d_newdata, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // Free GPU memory
    // hipFree(d_data);
    // hipFree(d_olddata);
    // hipFree(d_newdata);

    // Four edges
    #pragma omp parallel for shared(row_size, col_size, newdata, data, olddata, C, K, dt)
    for(int i = 1; i < row_size - 1; i++) {
        double P1 = data[(i + 1) * col_size] + data[(i - 1) * col_size] + data[i * col_size + 1] - 3 * data[i * col_size];
        double P2 = data[(i + 1) * col_size + col_size - 1] + data[(i - 1) * col_size + col_size - 1] +
                    data[i * col_size + col_size - 2] - 3 * data[i * col_size + col_size - 1];
        double P3 = data[col_size + i] + data[i + 1] + data[i - 1] - 3 * data[i];
        double P4 = data[(row_size - 2) * col_size + i] + data[(row_size - 1) * col_size + i + 1] +
                    data[(row_size - 1) * col_size + i - 1] - 3 * data[(row_size - 1) * col_size + i];
        newdata[i * col_size] = ( pow(C * dt, 2) * P1 * 2 + 4 * data[i * col_size] - olddata[i * col_size] *
                                    (2 - K * dt) ) / (2 + K * dt);
        newdata[i * col_size + col_size - 1] = ( pow(C * dt, 2) * P2 * 2 + 4 * data[i * col_size + col_size - 1] -
                                                olddata[i * col_size + col_size - 1] * (2 - K * dt) ) / (2 + K * dt);
        newdata[i] = ( pow(C * dt, 2) * P3 * 2 + 4 * data[i] - olddata[i] * (2 - K * dt) ) / (2 + K * dt);
        newdata[(row_size - 1) * col_size + i] = ( pow(C * dt, 2) * P4 * 2 + 4 * data[(row_size - 1) * col_size + i] -
                olddata[(row_size - 1) * col_size + i] * (2 - K * dt) ) / (2 + K * dt);
    }

    // Four corners
    double P1 = data[col_size] + data[1] - 2 * data[0];
    double P2 = data[col_size + col_size - 1] + data[col_size - 2] - 2 * data[col_size - 1];
    double P3 = data[(row_size - 2) * col_size] + data[(row_size - 1) * col_size + 1] - 2 * data[(row_size - 1) * col_size];
    double P4 = data[(row_size - 2) * col_size + col_size - 1] + data[(row_size - 1) * col_size + col_size - 2] - 2 *
                data[(row_size - 1) * col_size + col_size - 1];
    newdata[0] = ( pow(C * dt, 2) * P1 * 2 + 4 * data[0] - olddata[0] * (2 - K * dt) ) / (2 + K * dt);
    newdata[col_size - 1] = ( pow(C * dt, 2) * P2 * 2 + 4 * data[col_size - 1] - olddata[col_size - 1] * (2 - K * dt) ) /
                            (2 + K * dt);
    newdata[(row_size - 1) * col_size] = ( pow(C * dt, 2) * P3 * 2 + 4 * data[(row_size - 1) * col_size] - olddata[(row_size - 1)
                                            * col_size] * (2 - K * dt) ) / (2 + K * dt);
    newdata[(row_size - 1) * col_size + col_size - 1] = ( pow(C * dt, 2) * P4 * 2 +
            4 * data[(row_size - 1) * col_size + col_size - 1] - olddata[(row_size - 1) * col_size + col_size - 1] * (2 - K * dt) )
            / (2 + K * dt);

}
